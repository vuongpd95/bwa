#include "hip/hip_runtime.h"
/******************************************************************************
* PROGRAM: se_kernel
* PURPOSE: This is a collection of functions which is intended to optimize the
* 	speed of seed extension step in BWA MEM procedure.
*
*
* NAME: Vuong Pham-Duy.
*		College student.
*       Faculty of Computer Science and Engineering.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/
#include "se_kernel.h"

extern "C" void cuda_mem_chain2aln(const mem_opt_t *opt, const bntseq_t *bns, const uint8_t *pac, \
		int l_query, const uint8_t *query, const mem_chain_t *c, mem_alnreg_v *av);

/* CUDA support function */
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "[M::%s] GPUassert: %s %s %d\n", __func__, \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}

void print_mem_info()
{
	size_t free_byte;
	size_t total_byte;
	gpuErrchk(hipMemGetInfo(&free_byte, &total_byte));

	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;

	fprintf(stderr, "[M::%s] GPU memory usage: used = %.2f MB, free = %.2f MB, total = %.2f MB\n", \
		__func__, used_db/ONE_MBYTE, free_db/ONE_MBYTE, total_db/ONE_MBYTE);
}

/********************
 *** SW extension ***
 ********************/
int cuda_ksw_extend2(int qlen, const uint8_t *query, int tlen, const uint8_t *target, int m, const int8_t *mat, int o_del, int e_del, int o_ins, int e_ins, int w, int end_bonus, int zdrop, int h0, int *_qle, int *_tle, int *_gtle, int *_gscore, int *_max_off)
{
	eh_t *eh; // score array
	int8_t *qp; // query profile
	int i, j, k, oe_del = o_del + e_del, oe_ins = o_ins + e_ins, beg, end, max, max_i, max_j, max_ins, max_del, max_ie, gscore, max_off;
	assert(h0 > 0);
	// allocate memory
	qp = (int8_t*)malloc(qlen * m);
	eh = (eh_t*)calloc(qlen + 1, 8);
	// generate the query profile
	for (k = i = 0; k < m; ++k) {
		const int8_t *p = &mat[k * m];
		for (j = 0; j < qlen; ++j) qp[i++] = p[query[j]];
	}
	// fill the first row
	eh[0].h = h0; eh[1].h = h0 > oe_ins? h0 - oe_ins : 0;
	for (j = 2; j <= qlen && eh[j-1].h > e_ins; ++j)
		eh[j].h = eh[j-1].h - e_ins;
	// adjust $w if it is too large
	k = m * m;
	for (i = 0, max = 0; i < k; ++i) // get the max score
		max = max > mat[i]? max : mat[i];
	max_ins = (int)((double)(qlen * max + end_bonus - o_ins) / e_ins + 1.);
	max_ins = max_ins > 1? max_ins : 1;
	w = w < max_ins? w : max_ins;
	max_del = (int)((double)(qlen * max + end_bonus - o_del) / e_del + 1.);
	max_del = max_del > 1? max_del : 1;
	w = w < max_del? w : max_del; // TODO: is this necessary?
	// DP loop
	max = h0, max_i = max_j = -1; max_ie = -1, gscore = -1;
	max_off = 0;
	beg = 0, end = qlen;
	for (i = 0; LIKELY(i < tlen); ++i) {
		int t, f = 0, h1, m = 0, mj = -1;
		int8_t *q = &qp[target[i] * qlen];
		// apply the band and the constraint (if provided)
		if (beg < i - w) beg = i - w;
		if (end > i + w + 1) end = i + w + 1;
		if (end > qlen) end = qlen;
		// compute the first column
		if (beg == 0) {
			h1 = h0 - (o_del + e_del * (i + 1));
			if (h1 < 0) h1 = 0;
		} else h1 = 0;
		for (j = beg; LIKELY(j < end); ++j) {
			// At the beginning of the loop: eh[j] = { H(i-1,j-1), E(i,j) }, f = F(i,j) and h1 = H(i,j-1)
			// Similar to SSE2-SW, cells are computed in the following order:
			//   H(i,j)   = max{H(i-1,j-1)+S(i,j), E(i,j), F(i,j)}
			//   E(i+1,j) = max{H(i,j)-gapo, E(i,j)} - gape
			//   F(i,j+1) = max{H(i,j)-gapo, F(i,j)} - gape
			eh_t *p = &eh[j];
			int h, M = p->h, e = p->e; // get H(i-1,j-1) and E(i-1,j)
			p->h = h1;          // set H(i,j-1) for the next row
			M = M? M + q[j] : 0;// separating H and M to disallow a cigar like "100M3I3D20M"
			h = M > e? M : e;   // e and f are guaranteed to be non-negative, so h>=0 even if M<0
			h = h > f? h : f;
			h1 = h;             // save H(i,j) to h1 for the next column
			mj = m > h? mj : j; // record the position where max score is achieved
			m = m > h? m : h;   // m is stored at eh[mj+1]
			t = M - oe_del;
			t = t > 0? t : 0;
			e -= e_del;
			e = e > t? e : t;   // computed E(i+1,j)
			p->e = e;           // save E(i+1,j) for the next row
			t = M - oe_ins;
			t = t > 0? t : 0;
			f -= e_ins;
			f = f > t? f : t;   // computed F(i,j+1)
		}
		eh[end].h = h1; eh[end].e = 0;
		if (j == qlen) {
			max_ie = gscore > h1? max_ie : i;
			gscore = gscore > h1? gscore : h1;
		}
		if (m == 0) break;
		if (m > max) {
			max = m, max_i = i, max_j = mj;
			max_off = max_off > abs(mj - i)? max_off : abs(mj - i);
		} else if (zdrop > 0) {
			if (i - max_i > mj - max_j) {
				if (max - m - ((i - max_i) - (mj - max_j)) * e_del > zdrop) break;
			} else {
				if (max - m - ((mj - max_j) - (i - max_i)) * e_ins > zdrop) break;
			}
		}
		// update beg and end for the next round
		for (j = beg; LIKELY(j < end) && eh[j].h == 0 && eh[j].e == 0; ++j);
		beg = j;
		for (j = end; LIKELY(j >= beg) && eh[j].h == 0 && eh[j].e == 0; --j);
		end = j + 2 < qlen? j + 2 : qlen;
		//beg = 0; end = qlen; // uncomment this line for debugging
	}
	free(eh); free(qp);
	if (_qle) *_qle = max_j + 1;
	if (_tle) *_tle = max_i + 1;
	if (_gtle) *_gtle = max_ie + 1;
	if (_gscore) *_gscore = gscore;
	if (_max_off) *_max_off = max_off;
	return max;
}

static inline int cal_max_gap(const mem_opt_t *opt, int qlen)
{
	int l_del = (int)((double)(qlen * opt->a - opt->o_del) / opt->e_del + 1.);
	int l_ins = (int)((double)(qlen * opt->a - opt->o_ins) / opt->e_ins + 1.);
	int l = l_del > l_ins? l_del : l_ins;
	l = l > 1? l : 1;
	return l < opt->w<<1? l : opt->w<<1;
}

void cuda_mem_chain2aln(const mem_opt_t *opt, const bntseq_t *bns, const uint8_t *pac, int l_query, \
		const uint8_t *query, const mem_chain_t *c, mem_alnreg_v *av) {

	int i, k, rid, max_off[2], aw[2]; // aw: actual bandwidth used in extension
	int64_t l_pac = bns->l_pac, rmax[2], tmp, max = 0;
	const mem_seed_t *s;
	uint8_t *rseq = 0;
	uint64_t *srt;

	if (c->n == 0) return;
	// get the max possible span
	rmax[0] = l_pac << 1;
	rmax[1] = 0;
	for (i = 0; i < c->n; ++i) {
		int64_t b, e;
		const mem_seed_t *t = &c->seeds[i];
		b = t->rbeg - (t->qbeg + cal_max_gap(opt, t->qbeg));
		e = t->rbeg + t->len + ((l_query - t->qbeg - t->len) + cal_max_gap(opt, l_query - t->qbeg - t->len));
		rmax[0] = rmax[0] < b? rmax[0] : b;
		rmax[1] = rmax[1] > e? rmax[1] : e;
		if (t->len > max) max = t->len;
	}
	// rmax[0] = rmax[0] > 0? rmax[0] : 0;
	if(rmax[0] <= 0) rmax[0] = 0;
	// rmax[1] = rmax[1] < l_pac<<1? rmax[1] : l_pac<<1;
	if (rmax[1] >= (l_pac << 1)) rmax[1] = l_pac << 1;

	if (rmax[0] < l_pac && l_pac < rmax[1]) { // crossing the forward-reverse boundary; then choose one side
		// this works because all seeds are guaranteed to be on the same strand
		if (c->seeds[0].rbeg < l_pac) rmax[1] = l_pac;
		else rmax[0] = l_pac;
	}
	// retrieve the reference sequence
	rseq = bns_fetch_seq(bns, pac, &rmax[0], c->seeds[0].rbeg, &rmax[1], &rid);
	assert(c->rid == rid);

	srt = (uint64_t*)malloc(c->n * 8);
	for (i = 0; i < c->n; ++i)
		srt[i] = (uint64_t)c->seeds[i].score<<32 | i;
	ks_introsort_64(c->n, srt);
	for (k = c->n - 1; k >= 0; --k) {
		mem_alnreg_t *a;
		s = &c->seeds[(uint32_t)srt[k]];

		for (i = 0; i < av->n; ++i) { // test whether extension has been made before
			mem_alnreg_t *p = &av->a[i];
			int64_t rd;
			int qd, w, max_gap;
			if (s->rbeg < p->rb || s->rbeg + s->len > p->re || s->qbeg < p->qb || s->qbeg + s->len > p->qe)
				continue; // not fully contained
			if (s->len - p->seedlen0 > .1 * l_query) continue; // this seed may give a better alignment
			// qd: distance ahead of the seed on query; rd: on reference
			qd = s->qbeg - p->qb; rd = s->rbeg - p->rb;
			max_gap = cal_max_gap(opt, qd < rd? qd : rd); // the maximal gap allowed in regions ahead of the seed
			w = max_gap < p->w? max_gap : p->w; // bounded by the band width
			if (qd - rd < w && rd - qd < w) break; // the seed is "around" a previous hit
			// similar to the previous four lines, but this time we look at the region behind
			qd = p->qe - (s->qbeg + s->len); rd = p->re - (s->rbeg + s->len);
			max_gap = cal_max_gap(opt, qd < rd? qd : rd);
			w = max_gap < p->w? max_gap : p->w;
			if (qd - rd < w && rd - qd < w) break;
		}

		if (i < av->n) { // the seed is (almost) contained in an existing alignment; further testing is needed to confirm it is not leading to a different aln
			if (bwa_verbose >= 4)
				printf("** Seed(%d) [%ld;%ld,%ld] is almost contained in an existing alignment [%d,%d) <=> [%ld,%ld)\n",
					   k, (long)s->len, (long)s->qbeg, (long)s->rbeg, av->a[i].qb, av->a[i].qe, (long)av->a[i].rb, \
					   (long)av->a[i].re);
			for (i = k + 1; i < c->n; ++i) { // check overlapping seeds in the same chain
				const mem_seed_t *t;
				if (srt[i] == 0) continue;
				t = &c->seeds[(uint32_t)srt[i]];
				if (t->len < s->len * .95) continue; // only check overlapping if t is long enough;
				// TODO: more efficient by early stopping
				if (s->qbeg <= t->qbeg && s->qbeg + s->len - t->qbeg >= s->len>>2 && t->qbeg - s->qbeg != t->rbeg - s->rbeg)
					break;
				if (t->qbeg <= s->qbeg && t->qbeg + t->len - s->qbeg >= s->len>>2 && s->qbeg - t->qbeg != s->rbeg - t->rbeg)
					break;
			}
			if (i == c->n) { // no overlapping seeds; then skip extension
				srt[k] = 0; // mark that seed extension has not been performed
				continue;
			}
			if (bwa_verbose >= 4)
				printf("** Seed(%d) might lead to a different alignment even though it is contained. "
						"Extension will be performed.\n", k);
		}

		a = kv_pushp(mem_alnreg_t, *av);
		memset(a, 0, sizeof(mem_alnreg_t));
		a->w = aw[0] = aw[1] = opt->w;
		a->score = a->truesc = -1;
		a->rid = c->rid;

		if (bwa_verbose >= 4) err_printf("** ---> Extending from seed(%d) [%ld;%ld,%ld] @ %s <---\n", k, \
				(long)s->len, (long)s->qbeg, (long)s->rbeg, bns->anns[c->rid].name);
		if (s->qbeg) { // left extension
			uint8_t *rs, *qs;
			int qle, tle, gtle, gscore;
			qs = (uint8_t*)malloc(s->qbeg);
			for (i = 0; i < s->qbeg; ++i) qs[i] = query[s->qbeg - 1 - i];
			tmp = s->rbeg - rmax[0];
			rs = (uint8_t*)malloc(tmp);
			for (i = 0; i < tmp; ++i) rs[i] = rseq[tmp - 1 - i];
			for (i = 0; i < MAX_BAND_TRY; ++i) {
				int prev = a->score;
				aw[0] = opt->w << i;
				if (bwa_verbose >= 4) {
					int j;
					printf("*** Left ref:   ");
					for (j = 0; j < tmp; ++j)
						putchar("ACGTN"[(int)rs[j]]);
					putchar('\n');
					printf("*** Left query: ");
					for (j = 0; j < s->qbeg; ++j)
						putchar("ACGTN"[(int)qs[j]]);
					putchar('\n');
				}
				a->score = cuda_ksw_extend2(s->qbeg, qs, tmp, rs, 5, opt->mat, opt->o_del, opt->e_del, opt->o_ins, opt->e_ins, \
						aw[0], opt->pen_clip5, opt->zdrop, s->len * opt->a, &qle, &tle, &gtle, &gscore, &max_off[0]);
				if (bwa_verbose >= 4) {
					printf("*** Left extension: prev_score=%d; score=%d; bandwidth=%d; max_off_diagonal_dist=%d\n", \
							prev, a->score, aw[0], max_off[0]); fflush(stdout);
				}
				if (a->score == prev || max_off[0] < (aw[0]>>1) + (aw[0]>>2)) break;
			}
			// check whether we prefer to reach the end of the query
			if (gscore <= 0 || gscore <= a->score - opt->pen_clip5) { // local extension
				a->qb = s->qbeg - qle, a->rb = s->rbeg - tle;
				a->truesc = a->score;
			} else { // to-end extension
				a->qb = 0, a->rb = s->rbeg - gtle;
				a->truesc = gscore;
			}
			free(qs); free(rs);
		} else a->score = a->truesc = s->len * opt->a, a->qb = 0, a->rb = s->rbeg;

		if (s->qbeg + s->len != l_query) { // right extension
			int qle, tle, qe, re, gtle, gscore, sc0 = a->score;
			qe = s->qbeg + s->len;
			re = s->rbeg + s->len - rmax[0];
			assert(re >= 0);
			for (i = 0; i < MAX_BAND_TRY; ++i) {
				int prev = a->score;
				aw[1] = opt->w << i;
				if (bwa_verbose >= 4) {
					int j;
					printf("*** Right ref:   ");
					for (j = 0; j < rmax[1] - rmax[0] - re; ++j)
						putchar("ACGTN"[(int)rseq[re+j]]);
					putchar('\n');
					printf("*** Right query: ");
					for (j = 0; j < l_query - qe; ++j)
						putchar("ACGTN"[(int)query[qe+j]]);
					putchar('\n');
				}
				a->score = cuda_ksw_extend2(l_query - qe, query + qe, rmax[1] - rmax[0] - re, rseq + re, 5, opt->mat, \
						opt->o_del, opt->e_del, opt->o_ins, opt->e_ins, aw[1], opt->pen_clip3, opt->zdrop, sc0, \
						&qle, &tle, &gtle, &gscore, &max_off[1]);
				if (bwa_verbose >= 4) {
					printf("*** Right extension: prev_score=%d; score=%d; bandwidth=%d; max_off_diagonal_dist=%d\n", prev, a->score, aw[1], max_off[1]);
					fflush(stdout);
				}
				if (a->score == prev || max_off[1] < (aw[1]>>1) + (aw[1]>>2)) break;
			}
			// similar to the above
			if (gscore <= 0 || gscore <= a->score - opt->pen_clip3) { // local extension
				a->qe = qe + qle, a->re = rmax[0] + re + tle;
				a->truesc += a->score - sc0;
			} else { // to-end extension
				a->qe = l_query, a->re = rmax[0] + re + gtle;
				a->truesc += gscore - sc0;
			}
		} else a->qe = l_query, a->re = s->rbeg + s->len;
		if (bwa_verbose >= 4)
			printf("*** Added alignment region: [%d,%d) <=> [%ld,%ld); score=%d; {left,right}_bandwidth={%d,%d}\n", \
					a->qb, a->qe, (long)a->rb, (long)a->re, a->score, aw[0], aw[1]);

		// compute seedcov
		for (i = 0, a->seedcov = 0; i < c->n; ++i) {
			const mem_seed_t *t = &c->seeds[i];
			// seed fully contained
			if (t->qbeg >= a->qb && t->qbeg + t->len <= a->qe && t->rbeg >= a->rb && t->rbeg + t->len <= a->re)
				a->seedcov += t->len; // this is not very accurate, but for approx. mapQ, this is good enough
		}
		a->w = aw[0] > aw[1]? aw[0] : aw[1];
		a->seedlen0 = s->len;

		a->frac_rep = c->frac_rep;
	}
	free(srt); free(rseq);
}
