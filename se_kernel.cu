#include "hip/hip_runtime.h"
/******************************************************************************
* PROGRAM: se_kernel
* PURPOSE: This is a collection of functions which is used to optimize the 
* 	speed of seed extension step in BWA MEM procedure.
*
*
* NAME: Vuong Pham-Duy.
	College student.
*       Faculty of Computer Science and Engineering.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/
#include "se_kernel.h"

extern "C" void cuda_mem_process_seqs(const mem_opt_t *opt, const bwt_t *bwt, \
	const bntseq_t *bns, const uint8_t *pac, int64_t n_processed, int n, \
	bseq1_t *seqs, const mem_pestat_t *pes0);

mem_chain_v chain_mem_core(const mem_opt_t *opt, const bwt_t *bwt, \
	const bntseq_t *bns, const uint8_t *pac, int l_seq, char *seq, \
	void *buf);
mem_alnreg_v sort_dedup_patch_core(const mem_opt_t *opt, const bntseq_t *bns, \
	const uint8_t *pac, char *seq, mem_alnreg_v *regs);
void chn_mem(void *data, int i, int tid);
void mem_sort_dedup_patch(void *data, int i, int tid);
void cuda_seed_extension (const mem_opt_t *opt, const bntseq_t *bns, \
	const uint8_t *pac, int n, worker_t *w);

// Support functions
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}

// Constant variables, tested, usable and can apply to all const needed 
// variables.
__constant__ uint8_t d_pac;
__constant__ mem_opt_t d_opt;

void cuda_mem_process_seqs(const mem_opt_t *opt, const bwt_t *bwt, \
	const bntseq_t *bns, const uint8_t *pac, int64_t n_processed, int n, \
	bseq1_t *seqs, const mem_pestat_t *pes0)
{
	worker_t w;
	mem_pestat_t pes[4];
	double ctime, rtime;
	int i;
	ctime = cputime(); rtime = realtime();

	w.regs = (mem_alnreg_v*)malloc(n * sizeof(mem_alnreg_v));
	w.opt = opt;
	w.bwt = bwt;
	w.bns = bns;
	w.pac = pac;
	w.seqs = seqs;
	w.n_processed = n_processed;
	w.pes = &pes[0];

	if (opt->cuda_num_threads > 0) {
		w.aux = (smem_aux_t**)malloc(opt->n_threads * sizeof(smem_aux_t));
		for (i = 0; i < opt->n_threads; ++i)
			w.aux[i] = smem_aux_init();
		// Chaining mem
		kt_for(opt->n_threads, chn_mem, &w, \
			(opt->flag&MEM_F_PE)? n>>1 : n);
		
		// Perform seed extension
		cuda_seed_extension(opt, bns, pac, n, &w);		

		// mem sort and delete duplicated patch 
		kt_for(opt->n_threads, mem_sort_dedup_patch, &w, \
			(opt->flag&MEM_F_PE)? n>>1 : n);
		for (i = 0; i < opt->n_threads; ++i)
			smem_aux_destroy(w.aux[i]);
		free(w.aux);
		goto w2;
	}
	
	w.aux = (smem_aux_t**)malloc(opt->n_threads * sizeof(smem_aux_t));
	for (i = 0; i < opt->n_threads; ++i)
		w.aux[i] = smem_aux_init();
	// find mapping positions
	kt_for(opt->n_threads, worker1, &w, (opt->flag&MEM_F_PE)? n>>1 : n); 
	for (i = 0; i < opt->n_threads; ++i)
		smem_aux_destroy(w.aux[i]);
	free(w.aux);

w2:
	if (opt->flag&MEM_F_PE) { // infer insert sizes if not provided
		// if pes0 != NULL, set the insert-size distribution as pes0		
		if (pes0) {
			memcpy(pes, pes0, 4 * sizeof(mem_pestat_t));
		} else {
			// otherwise, infer the insert size distribution from 
			// data
			mem_pestat(opt, bns->l_pac, n, w.regs, pes);
		}
	}
	// generate alignment
	kt_for(opt->n_threads, worker2, &w, (opt->flag&MEM_F_PE)? n>>1 : n);
	free(w.regs);
	if (bwa_verbose >= 3)
		fprintf(stderr, "[M::%s] Processed %d reads in %.3f CPU sec, \
			%.3f real sec\n", __func__, n, cputime() - ctime, \
			realtime() - rtime);
}

void chn_mem(void *data, int i, int tid)
{
	worker_t *w = (worker_t*)data;
	if (!(w->opt->flag&MEM_F_PE)) {
		if (bwa_verbose >= 4) 
			printf("=====> Processing read '%s' <=====\n", \
			w->seqs[i].name);
		w->chns[i] = chain_mem_core(w->opt, w->bwt, w->bns, w->pac, \
			w->seqs[i].l_seq, w->seqs[i].seq, w->aux[tid]);
	} else {
		if (bwa_verbose >= 4) 
			printf("=====> Processing read '%s'/1 <=====\n", \
				w->seqs[i<<1|0].name);
		w->chns[i<<1|0] = chain_mem_core(w->opt, w->bwt, w->bns, \
			w->pac, w->seqs[i<<1|0].l_seq, w->seqs[i<<1|0].seq,\
			w->aux[tid]);
		if (bwa_verbose >= 4) 
			printf("=====> Processing read '%s'/2 <=====\n", \
				w->seqs[i<<1|1].name);
		w->chns[i<<1|1] = chain_mem_core(w->opt, w->bwt, w->bns, \
			w->pac, w->seqs[i<<1|1].l_seq, w->seqs[i<<1|1].seq, \
			w->aux[tid]);
	}
}

mem_chain_v chain_mem_core(const mem_opt_t *opt, const bwt_t *bwt, \
	const bntseq_t *bns, const uint8_t *pac, int l_seq, char *seq, \
	void *buf) {

	int i;
	mem_chain_v chn;

	for (i = 0; i < l_seq; ++i) 
		// convert to 2-bit encoding if we have not done so
		seq[i] = seq[i] < 4? seq[i] : nst_nt4_table[(int)seq[i]];

	chn = mem_chain(opt, bwt, bns, l_seq, (uint8_t*)seq, buf);
	chn.n = mem_chain_flt(opt, chn.n, chn.a);
	mem_flt_chained_seeds(opt, bns, pac, l_seq, (uint8_t*)seq, chn.n, chn.a);
	if (bwa_verbose >= 4) mem_print_chain(bns, &chn);
	
	return chn;	
}

void mem_sort_dedup_patch(void *data, int i, int tid) {
	worker_t *w = (worker_t*)data;
	if (!(w->opt->flag&MEM_F_PE)) {
		w->regs[i] = sort_dedup_patch_core(w->opt, w->bns, w->pac, \
			w->seqs[i].seq, &w->regs[i]);
	} else { 
		w->regs[i<<1|0] = sort_dedup_patch_core(w->opt, w->bns, \
			w->pac, w->seqs[i<<1|0].seq, &w->regs[i<<1|0]);

		w->regs[i<<1|1] = sort_dedup_patch_core(w->opt, w->bns, \
			w->pac, w->seqs[i<<1|1].seq, &w->regs[i<<1|1]);
	}
}

mem_alnreg_v sort_dedup_patch_core(const mem_opt_t *opt, const bntseq_t *bns, \
	const uint8_t *pac, char *seq, mem_alnreg_v *regs) {
	
	int i;
	regs->n = mem_sort_dedup_patch(opt, bns, pac, (uint8_t*)seq, regs->n, \
		regs->a);
	if (bwa_verbose >= 4) {
		err_printf("* %ld chains remain after removing duplicated chains\n", \
			regs->n);
		for (i = 0; i < regs->n; ++i) {
			mem_alnreg_t *p = &regs->a[i];
			printf("** %d, [%d,%d) <=> [%ld,%ld)\n", \
				p->score, p->qb, p->qe, (long)p->rb, (long)p->re);
		}
	}
	for (i = 0; i < regs->n; ++i) {
		mem_alnreg_t *p = &regs->a[i];
		if (p->rid >= 0 && bns->anns[p->rid].is_alt)
			p->is_alt = 1;
	}
	return (*regs);	
}

void cuda_seed_extension(const mem_opt_t *opt, const bntseq_t *bns, \
	const uint8_t *pac, int n, worker_t *w) {

	// Use w->seqs[...].l_seq + ...seqs[...].seq, w->regs, w->chns
	// Use opt, bns, pac
	int i, j;
	int n_a, n_seeds;

	bntann1_t *d_anns;

	int *l_seq, *dl_seq, sl_seq;
	uint8_t *seq, *d_seq;

	mem_chain_v *chns;
	flat_mem_chain_v *f_chns, *df_chns;
	flat_mem_chain_t *f_a, *df_a;
	mem_seed_t *seeds, *d_seeds;

	n_a = 0; 
	n_seeds = 0;
	sl_seq = 0;
	l_seq = (int*)malloc(n * sizeof(int));
	chns = w->chns;
	for(i = 0; i < n; i++) {
		l_seq[i] = seqs[i].l_seq;
		sl_seq += l_seq[i];
		n_a += chns[i].n;
		for(j = 0; j < chns[i].n; j++) {
			n_seeds += chns[i].a[j].n;
		}
	}
	seq = (uint8_t*)malloc(sl_seq * sizeof(uint8_t));
	int acc_seq;
	acc_seq = 0;
	for(i = 0; i < n; i++) {
		for(j = 0; j < l_seq[i]; j++) {
			seq[acc_seq + j] = w->seqs[i].seq[j];		
		}
		acc_seq += l_seq[i];
	}
	
	f_chns = (flat_mem_chain_v*)malloc(n * sizeof(flat_mem_chain_v));
	f_a = (flat_mem_chain_t*)malloc(n_a * sizeof(flat_mem_chain_t));
	seeds = (mem_seed_t*)malloc(n_seeds * sizeof(mem_seed_t));
	
	int acc_a, acc_seeds;
	acc_a = 0; acc_seeds = 0;

	for(i = 0; i < n; i++) {
		f_chns[i].n = chns[i].n;
		f_chns[i].m = chns[i].m;
		for(j = 0; j < chns[i].n; j++) {
			// int n, m, first, rid;
			// uint32_t w:29, kept:2, is_alt:1;
			// float frac_rep;
			// int64_t pos;
			mem_chain_t *tmp;
			tmp = &chns[i].a[j];
			f_a[acc_a].n = tmp->n;
			f_a[acc_a].m = tmp->m;
			f_a[acc_a].first = tmp->first;
			f_a[acc_a].rid = tmp->rid;
			f_a[acc_a].w = tmp->w;
			f_a[acc_a].kept = tmp->kept;
			f_a[acc_a].is_alt = tmp->is_alt;
			f_a[acc_a].frac_rep = tmp->frac_rep;
			f_a[acc_a].pos = tmp->pos;
			for(k = 0; k < chns[i].a[j].n; k++) {
				// int64_t rbeg;
				// int32_t qbeg, len;
				// int score;
				mem_seed_t *tmp0;
				tmp0 = &chns[i].a[j].seeds[k];
				seeds[acc_seeds].rbeg = tmp0->rbeg;
				seeds[acc_seeds].qbeg = tmp0->qbeg;
				seeds[acc_seeds].len = tmp0->len;
				seeds[acc_seeds].score = tmp0->score;
			}
			acc_seeds += chns[i].a[j].n;		
		}
		acc_a += chns[i].n;
	}

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(&d_pac), pac, sizeof(uint8_t)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(&d_opt), opt, sizeof(mem_opt_t)));

	gpuErrchk(hipMalloc(&d_anns, bns->n_seqs * sizeof(bntann1_t)));
	gpuErrchk(hipMalloc(&dl_seq, n * sizeof(int)));
	gpuErrchk(hipMalloc(&d_seq, sl_seq * sizeof(uint8_t)));

	gpuErrchk(hipMalloc(&df_chns, n * sizeof(flat_mem_chain_v)));
	gpuErrchk(hipMalloc(&df_a, n_a * sizeof(flat_mem_chain_t)));
	gpuErrchk(hipMalloc(&d_seeds, n_seeds * sizeof(mem_seed_t)));


	gpuErrchk(hipMemcpy(d_anns, bns->anns, bns->n_seqs * sizeof(bntann1_t), \
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dl_seq, l_seq, n * sizeof(int), \
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_seq, seq, sl_seq * sizeof(uint8_t), \
			hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(df_chns, f_chns, n * sizeof(flat_mem_chain_v), \
				hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(df_a, f_a, n_a * sizeof(flat_mem_chain_t), \
				hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_seeds, seeds, n_seeds * sizeof(mem_seed_t), \
				hipMemcpyHostToDevice));

	if (opt->flag & MEM_F_PE) {
						
	} else {

	}

	hipFree(d_anns);
	hipFree(dl_seq);
	hipFree(d_seq);
	hipFree(df_chns);
	hipFree(df_a);
	hipFree(d_seeds);
}

__global__ 
void extension_kernel(int64_t l_pac, int32_t n_seqs, bntann1_t *anns) {

}
