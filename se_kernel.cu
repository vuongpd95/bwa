#include "hip/hip_runtime.h"
/******************************************************************************
* PROGRAM: se_kernel
* PURPOSE: This is a collection of functions which is intended to optimize the
* 	speed of seed extension step in BWA MEM procedure.
*
*
* NAME: Vuong Pham-Duy.
*		College student.
*       Faculty of Computer Science and Engineering.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/
#include "se_kernel.h"

extern "C" void cuda_mem_chain2aln(const mem_opt_t *opt, const bntseq_t *bns, const uint8_t *pac, \
		int l_query, const uint8_t *query, const mem_chain_t *c, mem_alnreg_v *av);

/* CUDA support function */
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "[M::%s] GPUassert: %s %s %d\n", __func__, \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}

void print_mem_info()
{
	size_t free_byte;
	size_t total_byte;
	gpuErrchk(hipMemGetInfo(&free_byte, &total_byte));

	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;

	fprintf(stderr, "[M::%s] GPU memory usage: used = %.2f MB, free = %.2f MB, total = %.2f MB\n", \
		__func__, used_db/ONE_MBYTE, free_db/ONE_MBYTE, total_db/ONE_MBYTE);
}


static inline int cal_max_gap(const mem_opt_t *opt, int qlen)
{
	int l_del = (int)((double)(qlen * opt->a - opt->o_del) / opt->e_del + 1.);
	int l_ins = (int)((double)(qlen * opt->a - opt->o_ins) / opt->e_ins + 1.);
	int l = l_del > l_ins? l_del : l_ins;
	l = l > 1? l : 1;
	return l < opt->w<<1? l : opt->w<<1;
}

void cuda_mem_chain2aln(const mem_opt_t *opt, const bntseq_t *bns, const uint8_t *pac, int l_query, \
		const uint8_t *query, const mem_chain_t *c, mem_alnreg_v *av) {

	int i, k, rid, max_off[2], aw[2]; // aw: actual bandwidth used in extension
	int64_t l_pac = bns->l_pac, rmax[2], tmp, max = 0;
	const mem_seed_t *s;
	uint8_t *rseq = 0;
	uint64_t *srt;
	/** Query GPU device **/
	int device, proc_cnt;
	hipDeviceProp_t props;
	gpuErrchk(hipGetDevice(&device));
	gpuErrchk(hipGetDeviceProperties(&props, device));
	proc_cnt = props. multiProcessorCount;
	/** End querying **/
	if (c->n == 0) return;
	// get the max possible span
	rmax[0] = l_pac << 1;
	rmax[1] = 0;
	for (i = 0; i < c->n; ++i) {
		int64_t b, e;
		const mem_seed_t *t = &c->seeds[i];
		b = t->rbeg - (t->qbeg + cal_max_gap(opt, t->qbeg));
		e = t->rbeg + t->len + ((l_query - t->qbeg - t->len) + cal_max_gap(opt, l_query - t->qbeg - t->len));
		rmax[0] = rmax[0] < b? rmax[0] : b;
		rmax[1] = rmax[1] > e? rmax[1] : e;
		if (t->len > max) max = t->len;
	}
	// rmax[0] = rmax[0] > 0? rmax[0] : 0;
	if(rmax[0] <= 0) rmax[0] = 0;
	// rmax[1] = rmax[1] < l_pac<<1? rmax[1] : l_pac<<1;
	if (rmax[1] >= (l_pac << 1)) rmax[1] = l_pac << 1;

	if (rmax[0] < l_pac && l_pac < rmax[1]) { // crossing the forward-reverse boundary; then choose one side
		// this works because all seeds are guaranteed to be on the same strand
		if (c->seeds[0].rbeg < l_pac) rmax[1] = l_pac;
		else rmax[0] = l_pac;
	}
	// retrieve the reference sequence
	rseq = bns_fetch_seq(bns, pac, &rmax[0], c->seeds[0].rbeg, &rmax[1], &rid);
	assert(c->rid == rid);

	srt = (uint64_t*)malloc(c->n * 8);
	for (i = 0; i < c->n; ++i)
		srt[i] = (uint64_t)c->seeds[i].score<<32 | i;
	ks_introsort_64(c->n, srt);
	for (k = c->n - 1; k >= 0; --k) {
		mem_alnreg_t *a;
		s = &c->seeds[(uint32_t)srt[k]];
		/** Original
		for (i = 0; i < av->n; ++i) { // test whether extension has been made before
			mem_alnreg_t *p = &av->a[i];
			int64_t rd;
			int qd, w, max_gap;
			if (s->rbeg < p->rb || s->rbeg + s->len > p->re || s->qbeg < p->qb || s->qbeg + s->len > p->qe)
				continue; // not fully contained
			if (s->len - p->seedlen0 > .1 * l_query) continue; // this seed may give a better alignment
			// qd: distance ahead of the seed on query; rd: on reference
			qd = s->qbeg - p->qb; rd = s->rbeg - p->rb;
			max_gap = cal_max_gap(opt, qd < rd? qd : rd); // the maximal gap allowed in regions ahead of the seed
			w = max_gap < p->w? max_gap : p->w; // bounded by the band width
			if (qd - rd < w && rd - qd < w) break; // the seed is "around" a previous hit
			// similar to the previous four lines, but this time we look at the region behind
			qd = p->qe - (s->qbeg + s->len); rd = p->re - (s->rbeg + s->len);
			max_gap = cal_max_gap(opt, qd < rd? qd : rd);
			w = max_gap < p->w? max_gap : p->w;
			if (qd - rd < w && rd - qd < w) break;
		}
		**/
		/** CUDA ported **/
		int num_block;
		int tp_block;

		/** END of CUDA ported **/
		if (i < av->n) { // the seed is (almost) contained in an existing alignment; further testing is needed to confirm it is not leading to a different aln
			if (bwa_verbose >= 4)
				printf("** Seed(%d) [%ld;%ld,%ld] is almost contained in an existing alignment [%d,%d) <=> [%ld,%ld)\n",
					   k, (long)s->len, (long)s->qbeg, (long)s->rbeg, av->a[i].qb, av->a[i].qe, (long)av->a[i].rb, \
					   (long)av->a[i].re);
			for (i = k + 1; i < c->n; ++i) { // check overlapping seeds in the same chain
				const mem_seed_t *t;
				if (srt[i] == 0) continue;
				t = &c->seeds[(uint32_t)srt[i]];
				if (t->len < s->len * .95) continue; // only check overlapping if t is long enough;
				// TODO: more efficient by early stopping
				if (s->qbeg <= t->qbeg && s->qbeg + s->len - t->qbeg >= s->len>>2 && t->qbeg - s->qbeg != t->rbeg - s->rbeg)
					break;
				if (t->qbeg <= s->qbeg && t->qbeg + t->len - s->qbeg >= s->len>>2 && s->qbeg - t->qbeg != s->rbeg - t->rbeg)
					break;
			}
			if (i == c->n) { // no overlapping seeds; then skip extension
				srt[k] = 0; // mark that seed extension has not been performed
				continue;
			}
			if (bwa_verbose >= 4)
				printf("** Seed(%d) might lead to a different alignment even though it is contained. "
						"Extension will be performed.\n", k);
		}

		a = kv_pushp(mem_alnreg_t, *av);
		memset(a, 0, sizeof(mem_alnreg_t));
		a->w = aw[0] = aw[1] = opt->w;
		a->score = a->truesc = -1;
		a->rid = c->rid;

		if (bwa_verbose >= 4) err_printf("** ---> Extending from seed(%d) [%ld;%ld,%ld] @ %s <---\n", k, \
				(long)s->len, (long)s->qbeg, (long)s->rbeg, bns->anns[c->rid].name);
		if (s->qbeg) { // left extension
			uint8_t *rs, *qs;
			int qle, tle, gtle, gscore;
			qs = (uint8_t*)malloc(s->qbeg);
			for (i = 0; i < s->qbeg; ++i) qs[i] = query[s->qbeg - 1 - i];
			tmp = s->rbeg - rmax[0];
			rs = (uint8_t*)malloc(tmp);
			for (i = 0; i < tmp; ++i) rs[i] = rseq[tmp - 1 - i];
			for (i = 0; i < MAX_BAND_TRY; ++i) {
				int prev = a->score;
				aw[0] = opt->w << i;
				if (bwa_verbose >= 4) {
					int j;
					printf("*** Left ref:   ");
					for (j = 0; j < tmp; ++j)
						putchar("ACGTN"[(int)rs[j]]);
					putchar('\n');
					printf("*** Left query: ");
					for (j = 0; j < s->qbeg; ++j)
						putchar("ACGTN"[(int)qs[j]]);
					putchar('\n');
				}
				a->score = ksw_extend2(s->qbeg, qs, tmp, rs, 5, opt->mat, opt->o_del, opt->e_del, opt->o_ins, opt->e_ins, \
						aw[0], opt->pen_clip5, opt->zdrop, s->len * opt->a, &qle, &tle, &gtle, &gscore, &max_off[0]);
				if (bwa_verbose >= 4) {
					printf("*** Left extension: prev_score=%d; score=%d; bandwidth=%d; max_off_diagonal_dist=%d\n", \
							prev, a->score, aw[0], max_off[0]); fflush(stdout);
				}
				if (a->score == prev || max_off[0] < (aw[0]>>1) + (aw[0]>>2)) break;
			}
			// check whether we prefer to reach the end of the query
			if (gscore <= 0 || gscore <= a->score - opt->pen_clip5) { // local extension
				a->qb = s->qbeg - qle, a->rb = s->rbeg - tle;
				a->truesc = a->score;
			} else { // to-end extension
				a->qb = 0, a->rb = s->rbeg - gtle;
				a->truesc = gscore;
			}
			free(qs); free(rs);
		} else a->score = a->truesc = s->len * opt->a, a->qb = 0, a->rb = s->rbeg;

		if (s->qbeg + s->len != l_query) { // right extension
			int qle, tle, qe, re, gtle, gscore, sc0 = a->score;
			qe = s->qbeg + s->len;
			re = s->rbeg + s->len - rmax[0];
			assert(re >= 0);
			for (i = 0; i < MAX_BAND_TRY; ++i) {
				int prev = a->score;
				aw[1] = opt->w << i;
				if (bwa_verbose >= 4) {
					int j;
					printf("*** Right ref:   ");
					for (j = 0; j < rmax[1] - rmax[0] - re; ++j)
						putchar("ACGTN"[(int)rseq[re+j]]);
					putchar('\n');
					printf("*** Right query: ");
					for (j = 0; j < l_query - qe; ++j)
						putchar("ACGTN"[(int)query[qe+j]]);
					putchar('\n');
				}
				a->score = ksw_extend2(l_query - qe, query + qe, rmax[1] - rmax[0] - re, rseq + re, 5, opt->mat, \
						opt->o_del, opt->e_del, opt->o_ins, opt->e_ins, aw[1], opt->pen_clip3, opt->zdrop, sc0, \
						&qle, &tle, &gtle, &gscore, &max_off[1]);
				if (bwa_verbose >= 4) {
					printf("*** Right extension: prev_score=%d; score=%d; bandwidth=%d; max_off_diagonal_dist=%d\n", prev, a->score, aw[1], max_off[1]);
					fflush(stdout);
				}
				if (a->score == prev || max_off[1] < (aw[1]>>1) + (aw[1]>>2)) break;
			}
			// similar to the above
			if (gscore <= 0 || gscore <= a->score - opt->pen_clip3) { // local extension
				a->qe = qe + qle, a->re = rmax[0] + re + tle;
				a->truesc += a->score - sc0;
			} else { // to-end extension
				a->qe = l_query, a->re = rmax[0] + re + gtle;
				a->truesc += gscore - sc0;
			}
		} else a->qe = l_query, a->re = s->rbeg + s->len;
		if (bwa_verbose >= 4)
			printf("*** Added alignment region: [%d,%d) <=> [%ld,%ld); score=%d; {left,right}_bandwidth={%d,%d}\n", \
					a->qb, a->qe, (long)a->rb, (long)a->re, a->score, aw[0], aw[1]);

		// compute seedcov
		for (i = 0, a->seedcov = 0; i < c->n; ++i) {
			const mem_seed_t *t = &c->seeds[i];
			// seed fully contained
			if (t->qbeg >= a->qb && t->qbeg + t->len <= a->qe && t->rbeg >= a->rb && t->rbeg + t->len <= a->re)
				a->seedcov += t->len; // this is not very accurate, but for approx. mapQ, this is good enough
		}
		a->w = aw[0] > aw[1]? aw[0] : aw[1];
		a->seedlen0 = s->len;

		a->frac_rep = c->frac_rep;
	}
	free(srt); free(rseq);
}
